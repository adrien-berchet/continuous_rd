#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <sstream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/filesystem.hpp>
#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "class_parameters.hpp"
#include "dat_writer.hpp"

namespace ode = boost::numeric::odeint;


// Change this to float if your device does not support double computation
typedef double value_type;


#ifdef WITH_GPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
#else
typedef thrust::host_vector< value_type > state_type;
typedef thrust::host_vector< size_t > index_vector_type;
#endif


/**
 * \brief Define the dynamic system
*/
class rd_dynamics
{

public:

	struct sys_functor
	{
		template< class Tuple >
		__host__ __device__
		void operator()( Tuple t )  // This functor works on tuples of values
		{
			// Get current values
			const value_type u = thrust::get<0>(thrust::get<0>(t));
			const value_type v = thrust::get<1>(thrust::get<0>(t));
			const value_type w = thrust::get<2>(thrust::get<0>(t));

			// Get P sin(theta)
			const value_type P_sin_theta = thrust::get<3>(thrust::get<0>(t));

			// Get neighbors
			const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // Top neighbor for u
			const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // Top neighbor for v
			const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // Top neighbor for w
			const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // Bottom neighbor for u
			const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // Bottom neighbor for v
			const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // Bottom neighbor for w
			const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // Left neighbor for u
			const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // Left neighbor for v
			const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // Left neighbor for w
			const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // Right neighbor for u
			const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // Right neighbor for v
			const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // Right neighbor for w

			// Get cu, cv, cw
			const value_type cu = thrust::get<0>(thrust::get<5>(t));
			const value_type cv = thrust::get<1>(thrust::get<5>(t));
			const value_type cw = thrust::get<2>(thrust::get<5>(t));

			// Get Du, Dv, Dw
			const value_type Du = thrust::get<3>(thrust::get<5>(t));
			const value_type Dv = thrust::get<4>(thrust::get<5>(t));
			const value_type Dw = thrust::get<5>(thrust::get<5>(t));

			// Get c1, c2, ..., c9
			const value_type c1 = thrust::get<0>(thrust::get<6>(t));
			const value_type c2 = thrust::get<1>(thrust::get<6>(t));
			const value_type c3 = thrust::get<2>(thrust::get<6>(t));
			const value_type c4 = thrust::get<3>(thrust::get<6>(t));
			const value_type c5 = thrust::get<4>(thrust::get<6>(t));
			const value_type c6 = thrust::get<5>(thrust::get<6>(t));
			const value_type c7 = thrust::get<6>(thrust::get<6>(t));
			const value_type c8 = thrust::get<7>(thrust::get<6>(t));
			const value_type c9 = thrust::get<8>(thrust::get<6>(t));

			// Get Fmax, Gmax, Hmax
			const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
			const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
			const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

			// Compute each term for each component
			const value_type F_cond = c1 * v + c2 * w + c3;
			const value_type G_cond = c4 * u + c5 * w + c6;
			const value_type H_cond = c7 * u + c8 * v + c9;

			const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
			const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
			const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

			const value_type lapl_u = u_top + u_bot + u_left + u_right - 4 * u;
			const value_type lapl_v = v_top + v_bot + v_left + v_right - 4 * v;
			const value_type lapl_w = w_top + w_bot + w_left + w_right - 4 * w;

			// The dynamical equation
			thrust::get<0>(thrust::get<8>(t)) = F - cu * u + Du * lapl_u * P_sin_theta;
			thrust::get<1>(thrust::get<8>(t)) = G - cv * v + Dv * lapl_v * P_sin_theta;
			thrust::get<2>(thrust::get<8>(t)) = H - cw * w + Dw * lapl_w * P_sin_theta;
		}
	};

	rd_dynamics(
		const size_t &Nx_in, const size_t &Ny_in,
		const double &cu_in, const double &cv_in, const double &cw_in,
		const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
		const double &Du_in, const double &Dv_in, const double &Dw_in,
		const double &Fmax_in, const double &Gmax_in, const double &Hmax_in
	):
		N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ),
		cu(cu_in), cv(cv_in), cw(cw_in),
		c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
		Du(Du_in), Dv(Dv_in), Dw(Dw_in),
		Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
		top( 3 * N ), bot( 3 * N ), left( 3 * N ), right( 3 * N )
	{
		// Define neighbors
		thrust::counting_iterator<size_t> counter( 0 );

		// Top neighbors
		thrust::copy( counter , counter + (N - Nx) , top.begin() + Nx ); // u component
		thrust::copy( counter + N , counter + (2 * N - Nx) , top.begin() + N + Nx ); // v component
		thrust::copy( counter + 2 * N , counter + (3 * N - Nx) , top.begin() + 2 * N + Nx ); // w component
		thrust::copy( counter + (N - Nx), counter + N , top.begin() ); // u component
		thrust::copy( counter + (2 * N - Nx), counter + 2 * N , top.begin() + N ); // v component
		thrust::copy( counter + (3 * N - Nx), counter + 3 * N , top.begin() + 2 * N); // w component

		// Bottom neighbors
		thrust::copy( counter + Nx , counter + N , bot.begin() ); // u component
		thrust::copy( counter + N + Nx , counter + 2 * N , bot.begin() + N ); // v component
		thrust::copy( counter + 2 * N + Nx , counter + 3 * N , bot.begin() + 2 * N ); // w component
		thrust::copy( counter, counter + Nx , bot.begin() + N - Nx ); // u component
		thrust::copy( counter + N, counter + N + Nx , bot.begin() + 2 * N - Nx ); // V component
		thrust::copy( counter + 2 * N, counter + 2 * N + Nx , bot.begin() + 3 * N - Nx ); // w component

		// Left neighbors
		thrust::copy( counter , counter + 3 * N - 1 , left.begin() + 1 );

		// Right neighbors
		thrust::copy( counter + 1 , counter + 3 * N , right.begin() );

		// Adjust left and right neighbors on sides
		for (int i = 0; i < Ny; ++i)
		{
			left[i * Nx] = (i + 1) * Nx - 1; // u component
			right[(i + 1) * Nx - 1] = i * Nx; // u component
			left[N + i * Nx] = N + (i + 1) * Nx - 1; // v component
			right[N + (i + 1) * Nx - 1] = N + i * Nx; // v component
			left[2 * N + i * Nx] = 2 * N + (i + 1) * Nx - 1; // w component
			right[2 * N + (i + 1) * Nx - 1] = 2 * N + i * Nx; // w component
		}
	}

	void operator() ( const state_type &x , state_type &dxdt , const value_type )
	{
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() ,
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
					)
			) ),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N,
							x.end()
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), top.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), bot.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), left.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), right.end() )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.begin() + 3 * N)
					)
			) ),
			sys_functor()
		);
	}

	const index_vector_type& get_top() const {return this->top;}
	const index_vector_type& get_bot() const {return this->bot;}
	const index_vector_type& get_left() const {return this->left;}
	const index_vector_type& get_right() const {return this->right;}
	const size_t& get_N() const {return this->N;}

private:

	const size_t N, Nx, Ny;
	const double cu, cv, cw;
	const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
	const double Du, Dv, Dw;
	const double Fmax, Gmax, Hmax;
	index_vector_type top, bot, left, right;
};

/**
 * \brief Compute the max length of the file names
*/
template<typename T>
size_t number_length(T &tmax, T &dt)
{
	std::ostringstream tmp;
	double int_part;
	double decimal_part = std::modf(tmax, &int_part);
	tmp << int_part + dt;
	return tmp.str().size();
}

/**
 * \brief Compute the max length of the decimal part of the file names
*/
template<typename T>
size_t number_precision(T &dt)
{
	std::ostringstream tmp;
	tmp << dt;
	return std::max(size_t(3), tmp.str().size()) - 2;
}

/**
 * \brief Define the observer used to export the results
*/
struct observer
{
    const Parameters &params;
    const size_t N;
    const size_t filename_length;
    const size_t precision;

    observer( const Parameters &params_in, const size_t &N_in ) : params( params_in ), N( N_in ), filename_length( number_length(params.tmax, params.dt) ), precision( number_precision(params.dt) ) {}

    template< class State >
    void operator()( const State &state , value_type t )
    {
    	// TODO: use params.delta_obs to skip some exports if they are too close from each other

    	// Format file name (zero padding to ensure that the file are always correctly sorted)
		std::ostringstream filename;
		filename << std::fixed << std::setprecision(precision) << std::setw(filename_length) << std::setfill('0') << t;

		// Create file
        generic::DatWriter data_file(params.result_folder + "/results/" + filename.str() + ".dat");

		// Write header
        data_file.write_header(std::to_string(t), params.Nx, params.Ny, "x", "y", "u", "v", "w", "P_sin_theta");

        // Write data
        // TODO: This is the slowest part of the code, try to improve it either by using binary files or by improving the iterators (but I/O are probably the main limitation)
        int num=0;
        for(
        	auto i=thrust::make_zip_iterator(thrust::make_tuple(
				state.begin(),
				state.begin() + N,
				state.begin() + 2 * N,
				state.begin() + 3 * N
			) );
			i != thrust::make_zip_iterator(thrust::make_tuple(
				state.begin() + N,
				state.begin() + 2 * N,
				state.begin() + 3 * N,
				state.begin() + 4 * N
			) );
			++i
		)
        {
        	const double x = (num % params.Nx) * params.epsilon;
        	const double y = int(num / params.Nx) * params.epsilon;
        	data_file.write_row(x, y, thrust::get<0>(*i), thrust::get<1>(*i), thrust::get<2>(*i), thrust::get<3>(*i));
        	++num;
        }
    }
};

/**
 * \brief Random initialization
*/
void random_init(std::vector< value_type > &state, const Parameters &params)
{}

/**
 * \brief Gaussian initialization. Only used for validation.
*/
void gauss_init(std::vector< value_type > &state, const Parameters &params)
{
	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;

	const double &epsilon = params.epsilon;
	const double &sigma = params.gauss_std;

	const double x_center = (Nx / 2.0) * epsilon;
	const double y_center = (Ny / 2.0) * epsilon;

	std::cout<<"Gaussian initialization:"<<std::endl;
	std::cout<<"\t x_center = "<<x_center<<std::endl;
	std::cout<<"\t y_center = "<<y_center<<std::endl;
	std::cout<<"\t epsilon = "<<epsilon<<std::endl;
	std::cout<<"\t std = "<<sigma<<std::endl;

    int num=0;
    for(
    	auto i=thrust::make_zip_iterator(thrust::make_tuple(
			state.begin(),
			state.begin() + N,
			state.begin() + 2 * N
		) );
		i != thrust::make_zip_iterator(thrust::make_tuple(
			state.begin() + N,
			state.begin() + 2 * N,
			state.begin() + 3 * N
		) );
		++i
	)
    {
    	const double x = (num % Nx) * epsilon;
    	const double y = int(num / Nx) * epsilon;
		const double r = std::sqrt(std::pow(x - x_center, 2.) + std::pow(y - y_center, 2.));
		const double C = std::exp(-std::pow(r, 2.) / (2. * std::pow(sigma, 2.)));
    	thrust::get<0>(*i) = C;
    	thrust::get<1>(*i) = C;
    	thrust::get<2>(*i) = C;
    	++num;
    }
}

void export_neighbors(const rd_dynamics &sys, const Parameters &params)
{
	const size_t &N=sys.get_N();

	// Create file
    generic::DatWriter data_file(params.result_folder + "/neighbors.dat");

	// Write header
    data_file.write_header("Neighbors", params.Nx, params.Ny, "x", "y", "num", "top_u", "top_v", "top_w", "bot_u", "bot_v", "bot_w", "left_u", "left_v", "left_w", "right_u", "right_v", "right_w");

    // Write data
    int num=0;
    for(
    	auto i=thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin(),
						sys.get_top().begin() + N,
						sys.get_top().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() ,
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() ,
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() ,
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N
				) )
		) );
		i != thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin() + N ,
						sys.get_top().begin() + 2 * N ,
						sys.get_top().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N ,
						sys.get_bot().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N ,
						sys.get_left().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N ,
						sys.get_right().end()
				) )
		) );
		++i
	)
    {
    	const double x = (num % params.Nx) * params.epsilon;
    	const double y = int(num / params.Nx) * params.epsilon;
    	data_file.write_row(
    		x, y, num,
    		thrust::get<0>(thrust::get<0>(*i)), thrust::get<1>(thrust::get<0>(*i)), thrust::get<2>(thrust::get<0>(*i)),
    		thrust::get<0>(thrust::get<1>(*i)), thrust::get<1>(thrust::get<1>(*i)), thrust::get<2>(thrust::get<1>(*i)),
    		thrust::get<0>(thrust::get<2>(*i)), thrust::get<1>(thrust::get<2>(*i)), thrust::get<2>(thrust::get<2>(*i)),
    		thrust::get<0>(thrust::get<3>(*i)), thrust::get<1>(thrust::get<3>(*i)), thrust::get<2>(thrust::get<3>(*i))
		);
    	++num;
    }
}

std::vector<value_type> simulate_rd(Parameters &params)
{
	// Get values from parameters
	const double &cu=params.cu, &cv=params.cv, &cw=params.cw;
	const double &c1=params.c1, &c2=params.c2, &c3=params.c3, &c4=params.c4, &c5=params.c5, &c6=params.c6, &c7=params.c7, &c8=params.c8, &c9=params.c9;
	const double &Du=params.Du, &Dv=params.Dv, &Dw=params.Dw;
	const double &Fmax=params.Fmax, &Gmax=params.Gmax, &Hmax=params.Hmax;

	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;
	const value_type &dt = params.dt;

	// Create vectors of data: all variables are concatenated into one vector for simplicity
	// Create initial conditions and initial values on host
	std::vector< value_type > x_host( 4 * N, 0 );
    std::fill(x_host.begin() + 3 * N, x_host.end(), 1.0); // Set P sin(theta) = 1 everywhere
	if (params.gauss_std > 0)
	{
		gauss_init(x_host, params);
	}
	else
	{
		random_init(x_host, params);
	}

	// Copy to device
	state_type x = x_host;

	// Create stepper
	ode::runge_kutta4< state_type , value_type , state_type , value_type > stepper;

	// Create phase oscillator system function
	rd_dynamics sys(
		Nx, Ny,
		cu, cv, cw,
		c1, c2, c3, c4, c5, c6, c7, c8, c9,
		Du, Dv, Dw,
		Fmax, Gmax, Hmax
	);

	// Export neighbors
	if(params.export_neighbors)
	{
		export_neighbors(sys, params);
	}

	// Create observer
	observer obs(params, N);

	// Integrate
	// TODO: Add stoping criteria but Boost::ODEINT does not provide an easy way to do this. I think this should be done inside the observer to interrupt the integration when the criteria is satisfied. Another solution can be to just use do_step() manually.
	integrate_const( stepper , sys , x , 0.0 , params.tmax , dt , boost::ref(obs));

	// Export results
	thrust::copy( x.begin() , x.end() , x_host.begin() );
	return x_host;
}

int main( int argc , char* argv[] )
{
	// Define and read the parameters
	Parameters params;
	params.read(argc, argv);
	std::cout<<params<<std::endl;

	// Create folders in which the results will be stored
	if(!boost::filesystem::is_directory(params.result_folder))
	{
		boost::filesystem::create_directories(params.result_folder);
	}
	if(!boost::filesystem::is_directory(params.result_folder + "/results"))
	{
		boost::filesystem::create_directories(params.result_folder + "/results");
	}

	// Export parameters used for the current simulation
	params.write_parameters(params.result_folder + "/parameters_used.prm");

	// Run the simulation
	std::vector<value_type> result = simulate_rd(params);
}
