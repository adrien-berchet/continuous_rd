#include "hip/hip_runtime.h"
#define BOOST_LOG_DYN_LINK 1

#include <cmath>
#include <iostream>
#include <sstream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/filesystem.hpp>
#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "dat_writer.hpp"
#include "logger.hpp"

#include "class_parameters.hpp"

namespace ode = boost::numeric::odeint;


// Change this to float if your device does not support double computation
typedef double value_type;


#ifdef WITH_GPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
#else
typedef thrust::host_vector< value_type > state_type;
typedef thrust::host_vector< size_t > index_vector_type;
#endif


/**
 * \brief Compute X coordinate
*/
inline
double get_x(const size_t &it, const size_t &Nx, const double &epsilon)
{
	return (it % Nx) * epsilon;
}


/**
 * \brief Compute X coordinate
*/
inline
double get_y(const size_t &it, const size_t &Nx, const double &epsilon)
{
	return int(it / Nx) * epsilon;
}

/**
 * \brief Define the dynamic system
*/
class rd_dynamics
{

public:

	struct sys_functor
	{
		template< class Tuple >
		__host__ __device__
		void operator()( Tuple t )  // This functor works on tuples of values
		{
			// Get current values
			const value_type u = thrust::get<0>(thrust::get<0>(t));
			const value_type v = thrust::get<1>(thrust::get<0>(t));
			const value_type w = thrust::get<2>(thrust::get<0>(t));
			const double epsilon = thrust::get<3>(thrust::get<0>(t));

			// Get P sin(theta) for each direction
			const value_type P_sin_theta_top = thrust::get<0>(thrust::get<8>(t));
			const value_type P_sin_theta_bot = thrust::get<1>(thrust::get<8>(t));
			const value_type P_sin_theta_left = thrust::get<2>(thrust::get<8>(t));
			const value_type P_sin_theta_right = thrust::get<3>(thrust::get<8>(t));

			// Get neighbors
			const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // Top neighbor for u
			const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // Top neighbor for v
			const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // Top neighbor for w
			const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // Bottom neighbor for u
			const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // Bottom neighbor for v
			const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // Bottom neighbor for w
			const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // Left neighbor for u
			const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // Left neighbor for v
			const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // Left neighbor for w
			const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // Right neighbor for u
			const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // Right neighbor for v
			const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // Right neighbor for w

			// Get cu, cv, cw
			const value_type cu = thrust::get<0>(thrust::get<5>(t));
			const value_type cv = thrust::get<1>(thrust::get<5>(t));
			const value_type cw = thrust::get<2>(thrust::get<5>(t));

			// Get Du, Dv, Dw
			const value_type Du = thrust::get<3>(thrust::get<5>(t));
			const value_type Dv = thrust::get<4>(thrust::get<5>(t));
			const value_type Dw = thrust::get<5>(thrust::get<5>(t));

			// Get c1, c2, ..., c9
			const value_type c1 = thrust::get<0>(thrust::get<6>(t));
			const value_type c2 = thrust::get<1>(thrust::get<6>(t));
			const value_type c3 = thrust::get<2>(thrust::get<6>(t));
			const value_type c4 = thrust::get<3>(thrust::get<6>(t));
			const value_type c5 = thrust::get<4>(thrust::get<6>(t));
			const value_type c6 = thrust::get<5>(thrust::get<6>(t));
			const value_type c7 = thrust::get<6>(thrust::get<6>(t));
			const value_type c8 = thrust::get<7>(thrust::get<6>(t));
			const value_type c9 = thrust::get<8>(thrust::get<6>(t));

			// Get Fmax, Gmax, Hmax
			const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
			const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
			const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

			// Compute each term for each component
			const value_type F_cond = c1 * v + c2 * w + c3;
			const value_type G_cond = c4 * u + c5 * w + c6;
			const value_type H_cond = c7 * u + c8 * v + c9;

			const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
			const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
			const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

			const value_type lapl_u = std::pow(epsilon, -2.0) * (
				  P_sin_theta_top * (u_top - u)
				+ P_sin_theta_bot * (u_bot - u)
				+ P_sin_theta_left * (u_left - u)
				+ P_sin_theta_right * (u_right - u));
			const value_type lapl_v = std::pow(epsilon, -2.0) * (
				  P_sin_theta_top * (v_top - v)
				+ P_sin_theta_bot * (v_bot - v)
				+ P_sin_theta_left * (v_left - v)
				+ P_sin_theta_right * (v_right - v));
			const value_type lapl_w = std::pow(epsilon, -2.0) * (
				  P_sin_theta_top * (w_top - w)
				+ P_sin_theta_bot * (w_bot - w)
				+ P_sin_theta_left * (w_left - w)
				+ P_sin_theta_right * (w_right - w));

			// The dynamical equation
			thrust::get<0>(thrust::get<9>(t)) = F - cu * u + Du * lapl_u;
			thrust::get<1>(thrust::get<9>(t)) = G - cv * v + Dv * lapl_v;
			thrust::get<2>(thrust::get<9>(t)) = H - cw * w + Dw * lapl_w;
		}
	};

	rd_dynamics(
		const size_t &Nx_in, const size_t &Ny_in, const double &epsilon_in,
		const double &cu_in, const double &cv_in, const double &cw_in,
		const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
		const double &Du_in, const double &Dv_in, const double &Dw_in,
		const double &Fmax_in, const double &Gmax_in, const double &Hmax_in,
		std::vector< double > &Pxx_top_in, std::vector< double > &Pxx_bot_in, std::vector< double > &Pxx_left_in, std::vector< double > &Pxx_right_in
	):
		N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ), epsilon( epsilon_in ),
		cu(cu_in), cv(cv_in), cw(cw_in),
		c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
		Du(Du_in), Dv(Dv_in), Dw(Dw_in),
		Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
		top( 3 * N ), bot( 3 * N ), left( 3 * N ), right( 3 * N ),
		Pxx_top( Pxx_top_in ), Pxx_bot( Pxx_bot_in ), Pxx_left( Pxx_left_in ), Pxx_right( Pxx_right_in )
	{
		BOOST_LOG_TRIVIAL(debug) << "RD dynamics initialization";

		// Define neighbors
		thrust::counting_iterator<size_t> counter( 0 );

		// Top neighbors
		thrust::copy( counter , counter + (N - Nx) , top.begin() + Nx ); // u component
		thrust::copy( counter + N , counter + (2 * N - Nx) , top.begin() + N + Nx ); // v component
		thrust::copy( counter + 2 * N , counter + (3 * N - Nx) , top.begin() + 2 * N + Nx ); // w component
		thrust::copy( counter + (N - Nx), counter + N , top.begin() ); // u component
		thrust::copy( counter + (2 * N - Nx), counter + 2 * N , top.begin() + N ); // v component
		thrust::copy( counter + (3 * N - Nx), counter + 3 * N , top.begin() + 2 * N); // w component

		// Bottom neighbors
		thrust::copy( counter + Nx , counter + N , bot.begin() ); // u component
		thrust::copy( counter + N + Nx , counter + 2 * N , bot.begin() + N ); // v component
		thrust::copy( counter + 2 * N + Nx , counter + 3 * N , bot.begin() + 2 * N ); // w component
		thrust::copy( counter, counter + Nx , bot.begin() + N - Nx ); // u component
		thrust::copy( counter + N, counter + N + Nx , bot.begin() + 2 * N - Nx ); // V component
		thrust::copy( counter + 2 * N, counter + 2 * N + Nx , bot.begin() + 3 * N - Nx ); // w component

		// Left neighbors
		thrust::copy( counter , counter + 3 * N - 1 , left.begin() + 1 );

		// Right neighbors
		thrust::copy( counter + 1 , counter + 3 * N , right.begin() );

		// Adjust left and right neighbors on sides
		for (int i = 0; i < Ny; ++i)
		{
			left[i * Nx] = (i + 1) * Nx - 1; // u component
			right[(i + 1) * Nx - 1] = i * Nx; // u component
			left[N + i * Nx] = N + (i + 1) * Nx - 1; // v component
			right[N + (i + 1) * Nx - 1] = N + i * Nx; // v component
			left[2 * N + i * Nx] = 2 * N + (i + 1) * Nx - 1; // w component
			right[2 * N + (i + 1) * Nx - 1] = 2 * N + i * Nx; // w component
		}
	}

	void operator() ( const state_type &x , state_type &dxdt , const value_type &t)
	{
		BOOST_LOG_TRIVIAL(debug) << "Compute dynamics for t="<<t;
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() ,
							x.begin() + N,
							x.begin() + 2 * N,
							thrust::make_constant_iterator(epsilon)
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							Pxx_top.begin(),
							Pxx_bot.begin(),
							Pxx_left.begin(),
							Pxx_right.begin()
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
					)
			) ),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N,
							thrust::make_constant_iterator(epsilon)
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), top.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), bot.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), left.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), right.end() )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							Pxx_top.end(),
							Pxx_bot.end(),
							Pxx_left.end(),
							Pxx_right.end()
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.begin() + 3 * N)
					)
			) ),
			sys_functor()
		);
	}

	const index_vector_type& get_top() const {return this->top;}
	const index_vector_type& get_bot() const {return this->bot;}
	const index_vector_type& get_left() const {return this->left;}
	const index_vector_type& get_right() const {return this->right;}
	const size_t& get_N() const {return this->N;}

private:

	const size_t N, Nx, Ny;
	const double cu, cv, cw;
	const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
	const double Du, Dv, Dw;
	const double Fmax, Gmax, Hmax;
	const double epsilon;
	index_vector_type top, bot, left, right;

public:
	const state_type Pxx_top, Pxx_bot, Pxx_left, Pxx_right;
};

/**
 * \brief Compute the max length of the file names
*/
template<typename T>
size_t number_length(T &tmax, T &dt)
{
	std::ostringstream tmp;
	double int_part;
	double decimal_part = std::modf(tmax, &int_part);
	tmp << int_part + dt;
	return tmp.str().size();
}

/**
 * \brief Compute the max length of the decimal part of the file names
*/
template<typename T>
size_t number_precision(T &dt)
{
	std::ostringstream tmp;
	tmp << dt;
	return std::max(size_t(3), tmp.str().size()) - 2;
}

/**
 * \brief Define the observer used to export the results
*/
struct observer
{
	const Parameters &params;
	const size_t N;
	const size_t filename_length;
	const size_t precision;
	const state_type &Pxx_top, &Pxx_bot, &Pxx_left, &Pxx_right;
	double last_t;


	observer(
		const Parameters &params_in, const size_t &N_in,
		const state_type &Pxx_top_in,
		const state_type &Pxx_bot_in,
		const state_type &Pxx_left_in,
		const state_type &Pxx_right_in
	):
		params( params_in ), N( N_in ),
		filename_length( number_length(params.tmax, params.dt) ),
		precision( number_precision(params.dt) ),
		Pxx_top( Pxx_top_in ), Pxx_bot( Pxx_bot_in ), Pxx_left( Pxx_left_in ), Pxx_right( Pxx_right_in )
	{
		BOOST_LOG_TRIVIAL(debug) << "Observer initialization";
		last_t = -1;
	}

	template< class State >
	void operator()( const State &state , value_type t )
	{
		// TODO: use params.delta_obs to skip some exports if they are too close from each other
		if(t - last_t < params.delta_obs || t >= params.tmax - params.dt)
		{
			BOOST_LOG_TRIVIAL(debug) << "Skip export results for t="<<t;
			return;
		}
		else
		{
			last_t = t;
		}

		// Format file name (zero padding to ensure that the file are always correctly sorted)
		std::ostringstream filename;
		filename << std::fixed << std::setprecision(precision) << std::setw(filename_length) << std::setfill('0') << t << ".dat";

		BOOST_LOG_TRIVIAL(info) << "Export results for t="<<t<<" in "<<filename.str();

		// Create file
		generic::DatWriter data_file(params.result_folder + "/results/" + filename.str());

		// Write header
		data_file.write_header(std::to_string(t), params.Nx, params.Ny, "x", "y", "u", "v", "w", "P_sin_theta_top", "P_sin_theta_bot", "P_sin_theta_left", "P_sin_theta_right");

		// Write data
		// TODO: This is the slowest part of the code, try to improve it either by using binary files or by improving the iterators (but I/O are probably the main limitation)
		int num=0;
		for(
			auto i=thrust::make_zip_iterator(thrust::make_tuple(
				state.begin(),
				state.begin() + N,
				state.begin() + 2 * N,
				Pxx_top.begin(),
				Pxx_bot.begin(),
				Pxx_left.begin(),
				Pxx_right.begin()
			) );
			i != thrust::make_zip_iterator(thrust::make_tuple(
				state.begin() + N,
				state.begin() + 2 * N,
				state.begin() + 3 * N,
				Pxx_top.end(),
				Pxx_bot.end(),
				Pxx_left.end(),
				Pxx_right.end()
			) );
			++i
		)
		{
			data_file.write_row(
				get_x(num, params.Nx, params.epsilon),
				get_y(num, params.Nx, params.epsilon),
				thrust::get<0>(*i),
				thrust::get<1>(*i),
				thrust::get<2>(*i),
				thrust::get<3>(*i),
				thrust::get<4>(*i),
				thrust::get<5>(*i),
				thrust::get<6>(*i)
			);
			++num;
		}
	}
};

/**
 * \brief Random initialization
*/
void random_init(std::vector< value_type > &state, const Parameters &params, std::vector< double > &Pxx_top, std::vector< double > &Pxx_bot, std::vector< double > &Pxx_left, std::vector< double > &Pxx_right)
{
	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;

	const double &epsilon = params.epsilon;
	const double &S = params.S;

	BOOST_LOG_TRIVIAL(info) << "Random initialization:";
	BOOST_LOG_TRIVIAL(info) << "\t Nx = "<<Nx;
	BOOST_LOG_TRIVIAL(info) << "\t Ny = "<<Ny;
	BOOST_LOG_TRIVIAL(info) << "\t epsilon = "<<epsilon;
	BOOST_LOG_TRIVIAL(info) << "\t S = "<<S;

	int num=0;
	for(auto i=state.begin() + 3 * N; i != state.begin() + 4 * N; ++i)
	{
		const double x = get_x(num, Nx, epsilon);
		const double y = get_y(num, Nx, epsilon);
		// const double r = std::sqrt(std::pow(x - x_center, 2.) + std::pow(y - y_center, 2.));
		const double P_sin_theta = 0;
		*i = P_sin_theta;
		++num;
	}

}

/**
 * \brief Gaussian initialization. Only used for validation.
*/
void gauss_init(std::vector< value_type > &state, const Parameters &params, std::vector< double > &Pxx_top, std::vector< double > &Pxx_bot, std::vector< double > &Pxx_left, std::vector< double > &Pxx_right)
{
	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;

	const double &epsilon = params.epsilon;
	const double &sigma = params.gauss_std;

	const double x_center = (Nx / 2.0) * epsilon;
	const double y_center = (Ny / 2.0) * epsilon;

	BOOST_LOG_TRIVIAL(info) << "Gaussian initialization:";
	BOOST_LOG_TRIVIAL(info) << "\t Nx = "<<Nx;
	BOOST_LOG_TRIVIAL(info) << "\t Ny = "<<Ny;
	BOOST_LOG_TRIVIAL(info) << "\t x_center = "<<x_center;
	BOOST_LOG_TRIVIAL(info) << "\t y_center = "<<y_center;
	BOOST_LOG_TRIVIAL(info) << "\t epsilon = "<<epsilon;
	BOOST_LOG_TRIVIAL(info) << "\t std = "<<sigma;

	int num=0;
	for(
		auto i=thrust::make_zip_iterator(thrust::make_tuple(
			state.begin(),
			state.begin() + N,
			state.begin() + 2 * N
		) );
		i != thrust::make_zip_iterator(thrust::make_tuple(
			state.begin() + N,
			state.begin() + 2 * N,
			state.begin() + 3 * N
		) );
		++i
	)
	{
		const double x = get_x(num, Nx, epsilon);
		const double y = get_y(num, Nx, epsilon);
		const double r = std::sqrt(std::pow(x - x_center, 2.) + std::pow(y - y_center, 2.));
		const double C = std::exp(-std::pow(r, 2.) / (2. * std::pow(sigma, 2.)));
		thrust::get<0>(*i) = C;
		thrust::get<1>(*i) = C;
		thrust::get<2>(*i) = C;
		++num;
	}

	// Set P sin(theta) = 1 everywhere for all direction
	std::fill(Pxx_top.begin(), Pxx_top.end(), 1.0);
	std::fill(Pxx_bot.begin(), Pxx_bot.end(), 1.0);
	std::fill(Pxx_left.begin(), Pxx_left.end(), 1.0);
	std::fill(Pxx_right.begin(), Pxx_right.end(), 1.0);
}

/**
 * \brief Export neighbors indices used to compute the laplacian. Only used for validation.
*/
void export_neighbors(const rd_dynamics &sys, const Parameters &params)
{
	BOOST_LOG_TRIVIAL(info) << "Export neighbors";
	const size_t &N=sys.get_N();

	// Create file
	generic::DatWriter data_file(params.result_folder + "/neighbors.dat");

	// Write header
	data_file.write_header("Neighbors", params.Nx, params.Ny, "x", "y", "num", "top_u", "top_v", "top_w", "bot_u", "bot_v", "bot_w", "left_u", "left_v", "left_w", "right_u", "right_v", "right_w");

	// Write data
	int num=0;
	for(
		auto i=thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin(),
						sys.get_top().begin() + N,
						sys.get_top().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() ,
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() ,
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() ,
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N
				) )
		) );
		i != thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin() + N ,
						sys.get_top().begin() + 2 * N ,
						sys.get_top().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N ,
						sys.get_bot().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N ,
						sys.get_left().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N ,
						sys.get_right().end()
				) )
		) );
		++i
	)
	{
		const double x = get_x(num, params.Nx, params.epsilon);
		const double y = get_y(num, params.Nx, params.epsilon);
		data_file.write_row(
			x, y, num,
			thrust::get<0>(thrust::get<0>(*i)), thrust::get<1>(thrust::get<0>(*i)), thrust::get<2>(thrust::get<0>(*i)),
			thrust::get<0>(thrust::get<1>(*i)), thrust::get<1>(thrust::get<1>(*i)), thrust::get<2>(thrust::get<1>(*i)),
			thrust::get<0>(thrust::get<2>(*i)), thrust::get<1>(thrust::get<2>(*i)), thrust::get<2>(thrust::get<2>(*i)),
			thrust::get<0>(thrust::get<3>(*i)), thrust::get<1>(thrust::get<3>(*i)), thrust::get<2>(thrust::get<3>(*i))
		);
		++num;
	}
}

std::vector<value_type> simulate_rd(Parameters &params)
{
	// Get values from parameters
	const double &cu=params.cu, &cv=params.cv, &cw=params.cw;
	const double &c1=params.c1, &c2=params.c2, &c3=params.c3, &c4=params.c4, &c5=params.c5, &c6=params.c6, &c7=params.c7, &c8=params.c8, &c9=params.c9;
	const double &Du=params.Du, &Dv=params.Dv, &Dw=params.Dw;
	const double &Fmax=params.Fmax, &Gmax=params.Gmax, &Hmax=params.Hmax;

	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;
	const double &epsilon = params.epsilon;
	const value_type &dt = params.dt;

	// Create vectors of data: all variables are concatenated into one vector for simplicity
	// Create initial conditions and initial values on host
	std::vector< value_type > x_host( 3 * N, 0 );
	std::vector< double > Pxx_top( N, 1.0 ), Pxx_bot( N, 1.0 ), Pxx_left( N, 1.0 ), Pxx_right( N, 1.0 );
	if (params.gauss_std > 0)
	{
		gauss_init(x_host, params, Pxx_top, Pxx_bot, Pxx_left, Pxx_right);
	}
	else
	{
		random_init(x_host, params, Pxx_top, Pxx_bot, Pxx_left, Pxx_right);
	}

	// Copy to device
	state_type x = x_host;

	// Create stepper
	ode::runge_kutta4< state_type , value_type , state_type , value_type > stepper;

	// Create phase oscillator system function
	rd_dynamics sys(
		Nx, Ny, epsilon,
		cu, cv, cw,
		c1, c2, c3, c4, c5, c6, c7, c8, c9,
		Du, Dv, Dw,
		Fmax, Gmax, Hmax,
		Pxx_top, Pxx_bot, Pxx_left, Pxx_right
	);

	// Export neighbors
	if(params.export_neighbors)
	{
		export_neighbors(sys, params);
	}

	// Create observer
	observer obs(params, N, sys.Pxx_top, sys.Pxx_bot, sys.Pxx_left, sys.Pxx_right);

	// Integrate
	// TODO: Add stoping criteria but Boost::ODEINT does not provide an easy way to do this. I think this should be done inside the observer to interrupt the integration when the criteria is satisfied. Another solution can be to just use do_step() manually.
	integrate_const( stepper , sys , x , 0.0 , params.tmax , dt , boost::ref(obs));

	// Export results
	thrust::copy( x.begin() , x.end() , x_host.begin() );
	return x_host;
}

int main( int argc , char* argv[] )
{
	// Define and read the parameters
	Parameters params;
	params.read(argc, argv);
	std::cout<<params<<std::endl;

	// Initialize the logger
	generic::init_logger(params.log_level);

	// Create folders in which the results will be stored
	if(!boost::filesystem::is_directory(params.result_folder))
	{
		boost::filesystem::create_directories(params.result_folder);
	}
	if(!boost::filesystem::is_directory(params.result_folder + "/results"))
	{
		boost::filesystem::create_directories(params.result_folder + "/results");
	}

	// Export parameters used for the current simulation
	BOOST_LOG_TRIVIAL(info) << "Export parameters";
	params.write_parameters(params.result_folder + "/parameters_used.prm");

	// Run the simulation
	std::vector<value_type> result = simulate_rd(params);
}
