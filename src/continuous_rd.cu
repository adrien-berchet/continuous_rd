#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/filesystem.hpp>
#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "class_parameters.hpp"

using namespace std;

using namespace boost::numeric::odeint;


// change this to float if your device does not support double computation
typedef double value_type;


#ifdef WITH_GPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
#else
typedef thrust::host_vector< value_type > state_type;
typedef thrust::host_vector< size_t > index_vector_type;
#endif


class rd_dynamics
{

public:

	struct sys_functor
	{
		template< class Tuple >
		__host__ __device__
		void operator()( Tuple t )  // this functor works on tuples of values
		{
			// get current values
			const value_type u = thrust::get<0>(thrust::get<0>(t));
			const value_type v = thrust::get<1>(thrust::get<0>(t));
			const value_type w = thrust::get<2>(thrust::get<0>(t));

			// get P sin(theta)
			const value_type P_sin_theta = thrust::get<3>(thrust::get<0>(t));

			// get neighbors
			const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // top neighbor for u
			const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // top neighbor for v
			const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // top neighbor for w
			const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // bottom neighbor for u
			const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // bottom neighbor for v
			const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // bottom neighbor for w
			const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // left neighbor for u
			const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // left neighbor for v
			const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // left neighbor for w
			const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // right neighbor for u
			const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // right neighbor for v
			const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // right neighbor for w

			// get cu, cv, cw
			const value_type cu = thrust::get<0>(thrust::get<5>(t));
			const value_type cv = thrust::get<1>(thrust::get<5>(t));
			const value_type cw = thrust::get<2>(thrust::get<5>(t));

			// get Du, Dv, Dw
			const value_type Du = thrust::get<3>(thrust::get<5>(t));
			const value_type Dv = thrust::get<4>(thrust::get<5>(t));
			const value_type Dw = thrust::get<5>(thrust::get<5>(t));

			// get c1, c2, ..., c9
			const value_type c1 = thrust::get<0>(thrust::get<6>(t));
			const value_type c2 = thrust::get<1>(thrust::get<6>(t));
			const value_type c3 = thrust::get<2>(thrust::get<6>(t));
			const value_type c4 = thrust::get<3>(thrust::get<6>(t));
			const value_type c5 = thrust::get<4>(thrust::get<6>(t));
			const value_type c6 = thrust::get<5>(thrust::get<6>(t));
			const value_type c7 = thrust::get<6>(thrust::get<6>(t));
			const value_type c8 = thrust::get<7>(thrust::get<6>(t));
			const value_type c9 = thrust::get<8>(thrust::get<6>(t));

			// get Fmax, Gmax, Hmax
			const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
			const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
			const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

			// compute each term for each component
			const value_type F_cond = c1 * v + c2 * w + c3;
			const value_type G_cond = c4 * u + c5 * w + c6;
			const value_type H_cond = c7 * u + c8 * v + c9;

			const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
			const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
			const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

			const value_type lapl_u = u_top + u_bot + u_left + u_right - 4 * u;
			const value_type lapl_v = v_top + v_bot + v_left + v_right - 4 * v;
			const value_type lapl_w = w_top + w_bot + w_left + w_right - 4 * w;

			// the dynamical equation
			thrust::get<0>(thrust::get<8>(t)) = F - cu * u + Du * lapl_u * P_sin_theta;
			thrust::get<1>(thrust::get<8>(t)) = G - cv * v + Dv * lapl_v * P_sin_theta;
			thrust::get<2>(thrust::get<8>(t)) = H - cw * w + Dw * lapl_w * P_sin_theta;
		}
	};

	rd_dynamics(
		const state_type &init,
		const size_t &Nx_in, const size_t &Ny_in,
		const double &cu_in, const double &cv_in, const double &cw_in,
		const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
		const double &Du_in, const double &Dv_in, const double &Dw_in,
		const double &Fmax_in, const double &Gmax_in, const double &Hmax_in
	):
		state_n (init) , N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ),
		cu(cu_in), cv(cv_in), cw(cw_in),
		c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
		Du(Du_in), Dv(Dv_in), Dw(Dw_in),
		Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
		top( init.size() ), bot( init.size() ), left( init.size() ), right( init.size() )
	{
		// define neighbours
		thrust::counting_iterator<size_t> counter( 0 );

		// top neighbours
		thrust::copy( counter , counter+(N-Nx) , top.begin()+Nx );
		thrust::copy( counter+(N-Nx), counter+N , top.begin() );

		// bottom neighbours
		thrust::copy( counter+Nx , counter+N , bot.begin() );
		thrust::copy( counter, counter+Nx , bot.begin()+N-Nx );

		// left neighbours
		thrust::copy( counter , counter+N-1 , left.begin()+1 );

		// right neighbours
		thrust::copy( counter+1 , counter+N , right.begin() );

		// adjust left and right neighbours on sides
		for (int i = 0; i < Ny; ++i)
		{
			left[i * Nx] = i * Nx - 1;
			right[(i + 1) * Nx - 1] = i * Nx;
		}
	}

	void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
	{
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() ,
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , top.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , bot.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , left.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , right.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
					)
			) ),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N,
							x.end()
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , top.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , bot.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , left.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , right.end() )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.end())
					)
			) ),
			sys_functor()
		);
	}

private:

	const state_type &state_n;
	const size_t N, Nx, Ny;
	const double cu, cv, cw;
	const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
	const double Du, Dv, Dw;
	const double Fmax, Gmax, Hmax;
	index_vector_type top, bot, left, right;
};

struct observer
{
    value_type m_K_mean;
    size_t m_count;

    observer( void ) { }

    template< class State >
    void operator()( const State &x , value_type t )
    {
    	std::cout<<"t="<<t<<"s"<<std::endl;
    	thrust::copy( x.begin() , x.begin() + 10 , std::ostream_iterator< value_type >( std::cout , "\n" ) );
    	std::cout<<std::endl<<std::endl;
    }

};

std::vector<value_type> simulate_rd(Parameters &params)
{
	// get values from parameters
	const double cu=params.cu, cv=params.cv, cw=params.cw;
	const double c1=params.c1, c2=params.c2, c3=params.c3, c4=params.c4, c5=params.c5, c6=params.c6, c7=params.c7, c8=params.c8, c9=params.c9;
	const double Du=params.Du, Dv=params.Dv, Dw=params.Dw;
	const double Fmax=params.Fmax, Gmax=params.Gmax, Hmax=params.Hmax;
	const double P=params.P;

	const size_t Nx = params.Nx, Ny = params.Ny;
	const size_t N = Nx * Ny;
	const value_type dt = params.dt;

	// create vectors of data: all variables are concatenated into one vector for simplicity
	// create initial conditions and initial values on host:
	vector< value_type > x_host( 4 * N, 0 );
	vector< value_type > init_host( 4 * N, 0 );
	for( size_t i=0 ; i<(3 * N) ; ++i )
	{
		x_host[i] = 2.0 * drand48();
		init_host[i] = ( 4 * N - i ); // decreasing frequencies
	}
	for( size_t i=3 * N ; i<(4 * N) ; ++i )
	{
		x_host[i] = 1 + P;
		init_host[i] = 1 + P;
	}

	// copy to device
	state_type x = x_host;
	state_type init = init_host;

	// create stepper
	runge_kutta4< state_type , value_type , state_type , value_type > stepper;

	// create phase oscillator system function
	rd_dynamics sys(
		init,
		Nx, Ny,
		cu, cv, cw,
		c1, c2, c3, c4, c5, c6, c7, c8, c9,
		Du, Dv, Dw,
		Fmax, Gmax, Hmax
	);

	// create observer
	observer obs;

	// integrate
	integrate_const( stepper , sys , x , 0.0 , params.tmax , dt , boost::ref(obs));

	thrust::copy( x.begin() , x.end() , x_host.begin() );
	return x_host;
}

int main( int argc , char* argv[] )
{
	Parameters params;
	params.read(argc, argv);
	cout<<params<<endl;

	if(!boost::filesystem::is_directory(params.result_folder))
	{
		boost::filesystem::create_directories(params.result_folder);
	}
	params.write_parameters(params.result_folder + "/parameters_used.prm");

	vector<value_type> result = simulate_rd(params);

	// print some results
	std::copy( result.begin() , result.begin() + 10 , std::ostream_iterator< value_type >( std::cout , "\n" ) );
	std::cout << std::endl;
}
