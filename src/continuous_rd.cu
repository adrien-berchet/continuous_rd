#include "hip/hip_runtime.h"
/*
 Copyright 2011-2013 Mario Mulansky
 Copyright 2011 Karsten Ahnert
 Distributed under the Boost Software License, Version 1.0.
 (See accompanying file LICENSE_1_0.txt or
 copy at http://www.boost.org/LICENSE_1_0.txt)
 */

/*
 * This example shows how to use odeint on CUDA devices with thrust.
 * Note that we require at least Version 3.2 of the nVidia CUDA SDK
 * and the thrust library should be installed in the CUDA include
 * folder.
 *
 * As example we use a chain of phase oscillators with nearest neighbour
 * coupling, as described in:
 *
 * Avis H. Cohen, Philip J. Holmes and Richard H. Rand:
 * JOURNAL OF MATHEMATICAL BIOLOGY Volume 13, Number 3, 345-369,
 *
 */

#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

using namespace std;

using namespace boost::numeric::odeint;


//change this to float if your device does not support double computation
typedef double value_type;


//[ thrust_phase_chain_system
//change this to host_vector< ... > if you want to run on CPU

// typedef thrust::tuple< value_type, value_type > tuple_type;
// typedef thrust::device_vector< tuple_type > state_type;

typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;

//typedef thrust::host_vector< value_type > state_type;
//typedef thrust::host_vector< size_t > index_vector_type;

//<-
/*
 * This implements the rhs of the dynamical equation:
 * \phi'_0 = \omega_0 + sin( \phi_1 - \phi_0 )
 * \phi'_i  = \omega_i + sin( \phi_i+1 - \phi_i ) + sin( \phi_i - \phi_i-1 )
 * \phi'_N-1 = \omega_N-1 + sin( \phi_N-1 - \phi_N-2 )
 */
//->
class phase_oscillators
{

public:

    struct sys_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )  // this functor works on tuples of values
        {
            // // first, unpack the tuple into value, neighbors and omega
            // const value_type phi = thrust::get<0>(t);
            // const value_type phi_left = thrust::get<2>(thrust::get<1>(t));  // left neighbor
            // const value_type phi_right = thrust::get<3>(thrust::get<1>(t)); // right neighbor
            // const value_type omega = thrust::get<7>(t);
            // // the dynamical equation
            // thrust::get<8>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );

            // get component
            // const size_t component = thrust::get<9>(t);

            // get current values
            const value_type u = thrust::get<0>(thrust::get<0>(t));
            const value_type v = thrust::get<1>(thrust::get<0>(t));
            const value_type w = thrust::get<2>(thrust::get<0>(t));

            // get P sin(theta)
            const value_type P_sin_theta = thrust::get<3>(thrust::get<0>(t));

            // get neighbors
            const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // top neighbor for u
            const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // top neighbor for v
            const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // top neighbor for w
            const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // bottom neighbor for u
            const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // bottom neighbor for v
            const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // bottom neighbor for w
            const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // left neighbor for u
            const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // left neighbor for v
            const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // left neighbor for w
            const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // right neighbor for u
            const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // right neighbor for v
            const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // right neighbor for w

            // get cu, cv, cw
            const value_type cu = thrust::get<0>(thrust::get<5>(t));
            const value_type cv = thrust::get<1>(thrust::get<5>(t));
            const value_type cw = thrust::get<2>(thrust::get<5>(t));

            // get Du, Dv, Dw
            const value_type Du = thrust::get<3>(thrust::get<5>(t));
            const value_type Dv = thrust::get<4>(thrust::get<5>(t));
            const value_type Dw = thrust::get<5>(thrust::get<5>(t));

            // get c1, c2, ..., c9
            const value_type c1 = thrust::get<0>(thrust::get<6>(t));
            const value_type c2 = thrust::get<1>(thrust::get<6>(t));
            const value_type c3 = thrust::get<2>(thrust::get<6>(t));
            const value_type c4 = thrust::get<3>(thrust::get<6>(t));
            const value_type c5 = thrust::get<4>(thrust::get<6>(t));
            const value_type c6 = thrust::get<5>(thrust::get<6>(t));
            const value_type c7 = thrust::get<6>(thrust::get<6>(t));
            const value_type c8 = thrust::get<7>(thrust::get<6>(t));
            const value_type c9 = thrust::get<8>(thrust::get<6>(t));

            // get Fmax, Gmax, Hmax
            const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
            const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
            const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

            // get P
            // const value_type P = thrust::get<3>(thrust::get<7>(t));

            // compute each term for each component
            const value_type F_cond = c1 * v + c2 * w + c3;
            const value_type G_cond = c4 * u + c5 * w + c6;
            const value_type H_cond = c7 * u + c8 * v + c9;

            const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
            const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
            const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

            const value_type lapl_u = u_top + u_bot + u_left + u_right - 4 * u;
            const value_type lapl_v = v_top + v_bot + v_left + v_right - 4 * v;
            const value_type lapl_w = w_top + w_bot + w_left + w_right - 4 * w;

            // the dynamical equation
            thrust::get<0>(thrust::get<8>(t)) = F - cu * u + Du * lapl_u * P_sin_theta;
            thrust::get<1>(thrust::get<8>(t)) = G - cv * v + Dv * lapl_v * P_sin_theta;
            thrust::get<2>(thrust::get<8>(t)) = H - cw * w + Dw * lapl_w * P_sin_theta;
        }
    };

    phase_oscillators(
        const state_type &omega,
        const size_t &Nx_in, const size_t &Ny_in,
        const double &cu_in, const double &cv_in, const double &cw_in,
        const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
        const double &Du_in, const double &Dv_in, const double &Dw_in,
        const double &Fmax_in, const double &Gmax_in, const double &Hmax_in //,
        // const double &P_in
    ): m_omega( omega ) , m_N( omega.size() ) , m_prev( omega.size() ) , m_next( omega.size() ),
        state_n (omega) , N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ),
        cu(cu_in), cv(cv_in), cw(cw_in),
        c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
        Du(Du_in), Dv(Dv_in), Dw(Dw_in),
        Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
        // P(P_in),
        top( omega.size() ), bot( omega.size() ), left( omega.size() ), right( omega.size() )//,
        // component( 3 * Nx_in * Ny_in )
    {
        // build indices pointing to left and right neighbours
        thrust::counting_iterator<size_t> c( 0 );
        thrust::copy( c , c+m_N-1 , m_prev.begin()+1 );
        m_prev[0] = 0; // m_prev = { 0 , 0 , 1 , 2 , 3 , ... , N-1 }

        thrust::copy( c+1 , c+m_N , m_next.begin() );
        m_next[m_N-1] = m_N-1; // m_next = { 1 , 2 , 3 , ... , N-1 , N-1 }

        // define neighbours
        thrust::counting_iterator<size_t> counter( 0 );

        // top neighbours
        thrust::copy( counter , counter+(N-Nx) , top.begin()+Nx );
        thrust::copy( counter+(N-Nx), counter+N , top.begin() );

        // bottom neighbours
        thrust::copy( counter+Nx , counter+N , bot.begin() );
        thrust::copy( counter, counter+Nx , bot.begin()+N-Nx );

        // left neighbours
        thrust::copy( counter , counter+N-1 , left.begin()+1 );

        // right neighbours
        thrust::copy( counter+1 , counter+N , right.begin() );

        // adjust left and right neighbours on sides
        for (int i = 0; i < Ny; ++i)
        {
            left[i * Nx] = i * Nx - 1;
            right[(i + 1) * Nx - 1] = i * Nx;
        }

        // set component values
        // thrust::fill_n(component.begin(), N, 0);
        // thrust::fill_n(component.begin()+N, N, 1);
        // thrust::fill_n(component.begin()+2*N, N, 2);
    }

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            x.begin() ,
                            x.begin() + N,
                            x.begin() + 2 * N,
                            x.begin() + 3 * N
                    ) ),
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() , top.begin() ) ,
                            thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() , bot.begin() ) ,
                            thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() , left.begin() ) ,
                            thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() , right.begin() ) ,
                            thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N )
                    ) ) ,
                    thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
                    thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
                    thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
                    thrust::make_zip_iterator(
                        thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
                    )
            ) ),
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            x.begin() + N,
                            x.begin() + 2 * N,
                            x.begin() + 3 * N,
                            x.end()
                    ) ),
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N ),
                            thrust::make_permutation_iterator( x.end() , top.end() )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N ),
                            thrust::make_permutation_iterator( x.end() , bot.end() )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N ),
                            thrust::make_permutation_iterator( x.end() , left.end() )
                    ) ) ,
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
                            thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N ),
                            thrust::make_permutation_iterator( x.end() , right.end() )
                    ) ) ,
                    thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
                    thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
                    thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
                    thrust::make_zip_iterator(
                        thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.end())
                    )
            ) ),
            sys_functor()
        );
    }

private:

    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;

    const state_type &state_n;
    const size_t N, Nx, Ny;
    const double cu, cv, cw;
    const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
    const double Du, Dv, Dw;
    const double Fmax, Gmax, Hmax;
    // const double P;
    index_vector_type top, bot, left, right;
    // index_vector_type component;
};
//]

const size_t N = 8192;
const size_t Nx = 128, Ny = 64;
const value_type pi = 3.1415926535897932384626433832795029;
const value_type epsilon = 6.0 / ( N * N ); // should be < 8/N^2 to see phase locking
const value_type dt = 0.015;

int main( int arc , char* argv[] )
{
    // define constants
    const double cu=0.02, cv=0.025, cw=0.06;
    const double c1=-0.04, c2=-0.056, c3=0.382, c4=-0.05, c5=0, c6=0.25, c7=0.016, c8=-0.03, c9=0.24;
    const double Du=1.125, Dv=1.125, Dw=13.5;
    const double Fmax=0.5, Gmax=0.5, Hmax=0.5;
    const double P=0.00889;

    // create vcetors of data: all variables are concatenated into one vector for simplicity
    // create initial conditions and omegas on host:
    vector< value_type > x_host( 4 * N, 0 );
    vector< value_type > omega_host( 4 * N, 0 );
    for( size_t i=0 ; i<(3 * N) ; ++i )
    {
        x_host[i] = 2.0 * pi * drand48();
        omega_host[i] = ( 4 * N - i ) * epsilon; // decreasing frequencies
    }
    for( size_t i=3 * N ; i<(4 * N) ; ++i )
    {
        x_host[i] = 1 + P;
        omega_host[i] = 1 + P;
    }

    // copy to device
    state_type x = x_host;
    state_type omega = omega_host;

    // create stepper
    runge_kutta4< state_type , value_type , state_type , value_type > stepper;

    // create phase oscillator system function
    phase_oscillators sys(
        omega,
        Nx, Ny,
        cu, cv, cw,
        c1, c2, c3, c4, c5, c6, c7, c8, c9,
        Du, Dv, Dw,
        Fmax, Gmax, Hmax
    );

    // integrate
    integrate_const( stepper , sys , x , 0.0 , 10.0 , dt );

    // print some result
    thrust::copy( x.begin() , x.begin() + 10 , std::ostream_iterator< value_type >( std::cout , "\n" ) );
    std::cout << std::endl;
    //]
}

/*int main(int argc, const char *argv[])
{
    // Read parameters

    // Define variables
    int nx, ny;

    // Create initial state

    // Compute evolution

    // Export results
}*/
