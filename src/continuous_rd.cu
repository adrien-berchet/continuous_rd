#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <sstream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/filesystem.hpp>
#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "class_parameters.hpp"
#include "csv_writer.hpp"

using namespace std;

using namespace boost::numeric::odeint;


// Change this to float if your device does not support double computation
typedef double value_type;


#ifdef WITH_GPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
#else
typedef thrust::host_vector< value_type > state_type;
typedef thrust::host_vector< size_t > index_vector_type;
#endif


/**
 * \brief Define the dynamic system
*/
class rd_dynamics
{

public:

	struct sys_functor
	{
		template< class Tuple >
		__host__ __device__
		void operator()( Tuple t )  // This functor works on tuples of values
		{
			// Get current values
			const value_type u = thrust::get<0>(thrust::get<0>(t));
			const value_type v = thrust::get<1>(thrust::get<0>(t));
			const value_type w = thrust::get<2>(thrust::get<0>(t));

			// Get P sin(theta)
			const value_type P_sin_theta = thrust::get<3>(thrust::get<0>(t));

			// Get neighbors
			const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // Top neighbor for u
			const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // Top neighbor for v
			const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // Top neighbor for w
			const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // Bottom neighbor for u
			const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // Bottom neighbor for v
			const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // Bottom neighbor for w
			const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // Left neighbor for u
			const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // Left neighbor for v
			const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // Left neighbor for w
			const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // Right neighbor for u
			const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // Right neighbor for v
			const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // Right neighbor for w

			// Get cu, cv, cw
			const value_type cu = thrust::get<0>(thrust::get<5>(t));
			const value_type cv = thrust::get<1>(thrust::get<5>(t));
			const value_type cw = thrust::get<2>(thrust::get<5>(t));

			// Get Du, Dv, Dw
			const value_type Du = thrust::get<3>(thrust::get<5>(t));
			const value_type Dv = thrust::get<4>(thrust::get<5>(t));
			const value_type Dw = thrust::get<5>(thrust::get<5>(t));

			// Get c1, c2, ..., c9
			const value_type c1 = thrust::get<0>(thrust::get<6>(t));
			const value_type c2 = thrust::get<1>(thrust::get<6>(t));
			const value_type c3 = thrust::get<2>(thrust::get<6>(t));
			const value_type c4 = thrust::get<3>(thrust::get<6>(t));
			const value_type c5 = thrust::get<4>(thrust::get<6>(t));
			const value_type c6 = thrust::get<5>(thrust::get<6>(t));
			const value_type c7 = thrust::get<6>(thrust::get<6>(t));
			const value_type c8 = thrust::get<7>(thrust::get<6>(t));
			const value_type c9 = thrust::get<8>(thrust::get<6>(t));

			// Get Fmax, Gmax, Hmax
			const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
			const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
			const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

			// Compute each term for each component
			const value_type F_cond = c1 * v + c2 * w + c3;
			const value_type G_cond = c4 * u + c5 * w + c6;
			const value_type H_cond = c7 * u + c8 * v + c9;

			const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
			const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
			const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

			const value_type lapl_u = u_top + u_bot + u_left + u_right - 4 * u;
			const value_type lapl_v = v_top + v_bot + v_left + v_right - 4 * v;
			const value_type lapl_w = w_top + w_bot + w_left + w_right - 4 * w;

			// The dynamical equation
			thrust::get<0>(thrust::get<8>(t)) = F - cu * u + Du * lapl_u * P_sin_theta;
			thrust::get<1>(thrust::get<8>(t)) = G - cv * v + Dv * lapl_v * P_sin_theta;
			thrust::get<2>(thrust::get<8>(t)) = H - cw * w + Dw * lapl_w * P_sin_theta;
		}
	};

	rd_dynamics(
		const state_type &init,
		const size_t &Nx_in, const size_t &Ny_in,
		const double &cu_in, const double &cv_in, const double &cw_in,
		const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
		const double &Du_in, const double &Dv_in, const double &Dw_in,
		const double &Fmax_in, const double &Gmax_in, const double &Hmax_in
	):
		state_n (init) , N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ),
		cu(cu_in), cv(cv_in), cw(cw_in),
		c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
		Du(Du_in), Dv(Dv_in), Dw(Dw_in),
		Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
		top( init.size() ), bot( init.size() ), left( init.size() ), right( init.size() )
	{
		// Define neighbours
		thrust::counting_iterator<size_t> counter( 0 );

		// Top neighbours
		thrust::copy( counter , counter+(N-Nx) , top.begin()+Nx );
		thrust::copy( counter+(N-Nx), counter+N , top.begin() );

		// Bottom neighbours
		thrust::copy( counter+Nx , counter+N , bot.begin() );
		thrust::copy( counter, counter+Nx , bot.begin()+N-Nx );

		// Left neighbours
		thrust::copy( counter , counter+N-1 , left.begin()+1 );

		// Right neighbours
		thrust::copy( counter+1 , counter+N , right.begin() );

		// Adjust left and right neighbours on sides
		for (int i = 0; i < Ny; ++i)
		{
			left[i * Nx] = i * Nx - 1;
			right[(i + 1) * Nx - 1] = i * Nx;
		}
	}

	void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
	{
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() ,
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , top.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , bot.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , left.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() , right.begin() ) ,
							thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
					)
			) ),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N,
							x.end()
					) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , top.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , top.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , top.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , bot.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , bot.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , bot.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , left.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , left.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , left.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin() + N , right.begin() + N ) ,
							thrust::make_permutation_iterator( x.begin() + 2 * N , right.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.end() , right.end() )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.end())
					)
			) ),
			sys_functor()
		);
	}

private:

	const state_type &state_n;
	const size_t N, Nx, Ny;
	const double cu, cv, cw;
	const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
	const double Du, Dv, Dw;
	const double Fmax, Gmax, Hmax;
	index_vector_type top, bot, left, right;
};

/**
 * \brief Compute the max length of the file names
*/
template<typename T>
size_t number_length(T &tmax, T &dt)
{
	std::ostringstream tmp;
	double int_part;
	double decimal_part = std::modf(tmax, &int_part);
	tmp << int_part + dt;
	return tmp.str().size();
}

/**
 * \brief Compute the max length of the decimal part of the file names
*/
template<typename T>
size_t number_precision(T &dt)
{
	std::ostringstream tmp;
	tmp << dt;
	return std::max(size_t(3), tmp.str().size()) - 2;
}

/**
 * \brief Define the observer used to export the results
*/
struct observer
{
    const Parameters &params;
    const size_t N;
    const size_t filename_length;
    const size_t precision;

    observer( const Parameters &params_in, const size_t &N_in ) : params( params_in ), N( N_in ), filename_length( number_length(params.tmax, params.dt) ), precision( number_precision(params.dt) ) {}

    template< class State >
    void operator()( const State &state , value_type t )
    {
    	// Format file name (zero padding to ensure that the file are always correctly sorted)
		std::ostringstream filename;
		filename << std::fixed << std::setprecision(precision) << std::setw(filename_length) << std::setfill('0') << t;

		// Create file
        generic::CsvWriter csv_file(params.result_folder + "/results/" + filename.str() + ".dat");

		// Write header
        csv_file.write_header("x", "y", "u", "v", "w");

        // Write data
        // TODO: This is the slowest part of the code, try to improve it (I/O are probably limitating but it may be possible to improve the iterators)
        int num=0;
        for(
        	auto i=thrust::make_zip_iterator(thrust::make_tuple(
				state.begin(),
				state.begin() + N,
				state.begin() + 2 * N
			) );
			i != thrust::make_zip_iterator(thrust::make_tuple(
				state.begin() + N,
				state.begin() + 2 * N,
				state.begin() + 3 * N
			) );
			++i
		)
        {
        	const double x = (num % params.Nx) * params.epsilon;
        	const double y = int(num / params.Nx) * params.epsilon;
        	csv_file.write_row(x, y, thrust::get<0>(*i), thrust::get<1>(*i), thrust::get<2>(*i));
        	++num;
        }
    }

};

std::vector<value_type> simulate_rd(Parameters &params)
{
	// Get values from parameters
	const double &cu=params.cu, &cv=params.cv, &cw=params.cw;
	const double &c1=params.c1, &c2=params.c2, &c3=params.c3, &c4=params.c4, &c5=params.c5, &c6=params.c6, &c7=params.c7, &c8=params.c8, &c9=params.c9;
	const double &Du=params.Du, &Dv=params.Dv, &Dw=params.Dw;
	const double &Fmax=params.Fmax, &Gmax=params.Gmax, &Hmax=params.Hmax;
	const double &P=params.P;

	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;
	const value_type &dt = params.dt;

	// Create vectors of data: all variables are concatenated into one vector for simplicity
	// Create initial conditions and initial values on host
	vector< value_type > x_host( 4 * N, 0 );
	vector< value_type > init_host( 4 * N, 0 );
	for( size_t i=0 ; i<(3 * N) ; ++i )
	{
		x_host[i] = 2.0 * drand48();
		init_host[i] = ( 4 * N - i ); // decreasing frequencies
	}
	for( size_t i=3 * N ; i<(4 * N) ; ++i )
	{
		x_host[i] = 1 + P;
		init_host[i] = 1 + P;
	}

	// Copy to device
	state_type x = x_host;
	state_type init = init_host;

	// Create stepper
	runge_kutta4< state_type , value_type , state_type , value_type > stepper;

	// Create phase oscillator system function
	rd_dynamics sys(
		init,
		Nx, Ny,
		cu, cv, cw,
		c1, c2, c3, c4, c5, c6, c7, c8, c9,
		Du, Dv, Dw,
		Fmax, Gmax, Hmax
	);

	// Create observer
	observer obs(params, N);

	// Integrate
	integrate_const( stepper , sys , x , 0.0 , params.tmax , dt , boost::ref(obs));

	// Export results
	thrust::copy( x.begin() , x.end() , x_host.begin() );
	return x_host;
}

int main( int argc , char* argv[] )
{
	// Define and read the parameters
	Parameters params;
	params.read(argc, argv);
	cout<<params<<endl;

	// Create folders in which the results will be stored
	if(!boost::filesystem::is_directory(params.result_folder))
	{
		boost::filesystem::create_directories(params.result_folder);
	}
	if(!boost::filesystem::is_directory(params.result_folder + "/results"))
	{
		boost::filesystem::create_directories(params.result_folder + "/results");
	}
	params.write_parameters(params.result_folder + "/parameters_used.prm");

	// Run the simulation
	vector<value_type> result = simulate_rd(params);
}
