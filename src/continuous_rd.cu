#include "hip/hip_runtime.h"
#define BOOST_LOG_DYN_LINK 1

#include <algorithm>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <random>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/filesystem.hpp>
#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include "dat_writer.hpp"
#include "logger.hpp"

#include "class_parameters.hpp"
#include "hex_2d_lattice.hpp"

namespace ode = boost::numeric::odeint;

const std::tuple<double, double, double> black(6.85, 0.09, 4.75);
const std::tuple<double, double, double> green(0.05, 5.35, 0.09);


// Change this to float if your device does not support double computation
typedef double value_type;


typedef thrust::host_vector< value_type > host_state_type;
typedef thrust::host_vector< int > host_int_vector_type;
typedef thrust::host_vector< size_t > host_index_vector_type;

typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;


/**
 * \brief Compute X coordinate
*/
inline
double get_x(const size_t &it, const size_t &Nx, const double &epsilon)
{
	return (it % Nx) * epsilon;
}


/**
 * \brief Compute X coordinate
*/
inline
double get_y(const size_t &it, const size_t &Nx, const double &epsilon)
{
	return int(it / Nx) * epsilon;
}

/**
 * \brief Define the dynamic system
*/
class rd_dynamics
{

public:

	struct sys_functor
	{
		template< class Tuple >
		__host__ __device__
		void operator()( Tuple t )  // This functor works on tuples of values
		{
			// Get current values
			const value_type u = thrust::get<0>(thrust::get<0>(t));
			const value_type v = thrust::get<1>(thrust::get<0>(t));
			const value_type w = thrust::get<2>(thrust::get<0>(t));
			const double epsilon = thrust::get<3>(thrust::get<0>(t));

			// Get P sin(theta) for each direction
			const value_type P_sin_theta_top = thrust::get<0>(thrust::get<8>(t));
			const value_type P_sin_theta_bot = thrust::get<1>(thrust::get<8>(t));
			const value_type P_sin_theta_left = thrust::get<2>(thrust::get<8>(t));
			const value_type P_sin_theta_right = thrust::get<3>(thrust::get<8>(t));

			// Get neighbors
			const value_type u_top = thrust::get<0>(thrust::get<1>(t));  // Top neighbor for u
			const value_type v_top = thrust::get<1>(thrust::get<1>(t));  // Top neighbor for v
			const value_type w_top = thrust::get<2>(thrust::get<1>(t));  // Top neighbor for w
			const value_type u_bot = thrust::get<0>(thrust::get<2>(t));  // Bottom neighbor for u
			const value_type v_bot = thrust::get<1>(thrust::get<2>(t));  // Bottom neighbor for v
			const value_type w_bot = thrust::get<2>(thrust::get<2>(t));  // Bottom neighbor for w
			const value_type u_left = thrust::get<0>(thrust::get<3>(t));  // Left neighbor for u
			const value_type v_left = thrust::get<1>(thrust::get<3>(t));  // Left neighbor for v
			const value_type w_left = thrust::get<2>(thrust::get<3>(t));  // Left neighbor for w
			const value_type u_right = thrust::get<0>(thrust::get<4>(t)); // Right neighbor for u
			const value_type v_right = thrust::get<1>(thrust::get<4>(t)); // Right neighbor for v
			const value_type w_right = thrust::get<2>(thrust::get<4>(t)); // Right neighbor for w

			// Get cu, cv, cw
			const value_type cu = thrust::get<0>(thrust::get<5>(t));
			const value_type cv = thrust::get<1>(thrust::get<5>(t));
			const value_type cw = thrust::get<2>(thrust::get<5>(t));

			// Get Du, Dv, Dw
			const value_type Du = thrust::get<3>(thrust::get<5>(t));
			const value_type Dv = thrust::get<4>(thrust::get<5>(t));
			const value_type Dw = thrust::get<5>(thrust::get<5>(t));

			// Get c1, c2, ..., c9
			const value_type c1 = thrust::get<0>(thrust::get<6>(t));
			const value_type c2 = thrust::get<1>(thrust::get<6>(t));
			const value_type c3 = thrust::get<2>(thrust::get<6>(t));
			const value_type c4 = thrust::get<3>(thrust::get<6>(t));
			const value_type c5 = thrust::get<4>(thrust::get<6>(t));
			const value_type c6 = thrust::get<5>(thrust::get<6>(t));
			const value_type c7 = thrust::get<6>(thrust::get<6>(t));
			const value_type c8 = thrust::get<7>(thrust::get<6>(t));
			const value_type c9 = thrust::get<8>(thrust::get<6>(t));

			// Get Fmax, Gmax, Hmax
			const value_type Fmax = thrust::get<0>(thrust::get<7>(t));
			const value_type Gmax = thrust::get<1>(thrust::get<7>(t));
			const value_type Hmax = thrust::get<2>(thrust::get<7>(t));

			// Compute each term for each component
			const value_type F_cond = c1 * v + c2 * w + c3;
			const value_type G_cond = c4 * u + c5 * w + c6;
			const value_type H_cond = c7 * u + c8 * v + c9;

			const value_type F = thrust::max(0.0, thrust::min(Fmax, F_cond));
			const value_type G = thrust::max(0.0, thrust::min(Gmax, G_cond));
			const value_type H = thrust::max(0.0, thrust::min(Hmax, H_cond));

			const value_type lapl_u = (
				  P_sin_theta_top * (u_top - u)
				+ P_sin_theta_bot * (u_bot - u)
				+ P_sin_theta_left * (u_left - u)
				+ P_sin_theta_right * (u_right - u));
			const value_type lapl_v = (
				  P_sin_theta_top * (v_top - v)
				+ P_sin_theta_bot * (v_bot - v)
				+ P_sin_theta_left * (v_left - v)
				+ P_sin_theta_right * (v_right - v));
			const value_type lapl_w = (
				  P_sin_theta_top * (w_top - w)
				+ P_sin_theta_bot * (w_bot - w)
				+ P_sin_theta_left * (w_left - w)
				+ P_sin_theta_right * (w_right - w));

			// The dynamical equation
			thrust::get<0>(thrust::get<9>(t)) = F - cu * u + Du * lapl_u;
			thrust::get<1>(thrust::get<9>(t)) = G - cv * v + Dv * lapl_v;
			thrust::get<2>(thrust::get<9>(t)) = H - cw * w + Dw * lapl_w;
		}
	};

	rd_dynamics(
		const size_t &Nx_in, const size_t &Ny_in, const double &epsilon_in,
		const double &cu_in, const double &cv_in, const double &cw_in,
		const double &c1_in, const double &c2_in, const double &c3_in, const double &c4_in, const double &c5_in, const double &c6_in, const double &c7_in, const double &c8_in, const double &c9_in,
		const double &Du_in, const double &Dv_in, const double &Dw_in,
		const double &Fmax_in, const double &Gmax_in, const double &Hmax_in,
		host_state_type &Pxx_top_in, host_state_type &Pxx_bot_in, host_state_type &Pxx_left_in, host_state_type &Pxx_right_in
	):
		N ( Nx_in * Ny_in ), Nx( Nx_in ), Ny ( Ny_in ), epsilon( epsilon_in ),
		cu(cu_in), cv(cv_in), cw(cw_in),
		c1(c1_in), c2(c2_in), c3(c3_in), c4(c4_in), c5(c5_in), c6(c6_in), c7(c7_in), c8(c8_in), c9(c9_in),
		Du(Du_in), Dv(Dv_in), Dw(Dw_in),
		Fmax(Fmax_in), Gmax(Gmax_in), Hmax(Hmax_in),
		top( 3 * N ), bot( 3 * N ), left( 3 * N ), right( 3 * N ),
		Pxx_top( Pxx_top_in ), Pxx_bot( Pxx_bot_in ), Pxx_left( Pxx_left_in ), Pxx_right( Pxx_right_in )
	{
		BOOST_LOG_TRIVIAL(debug) << "RD dynamics initialization";

		// Define neighbors
		thrust::counting_iterator<size_t> counter( 0 );

		// Top neighbors
		thrust::copy( counter , counter + (N - Nx) , top.begin() + Nx ); // u component
		thrust::copy( counter + N , counter + (2 * N - Nx) , top.begin() + N + Nx ); // v component
		thrust::copy( counter + 2 * N , counter + (3 * N - Nx) , top.begin() + 2 * N + Nx ); // w component
		thrust::copy( counter + (N - Nx), counter + N , top.begin() ); // u component
		thrust::copy( counter + (2 * N - Nx), counter + 2 * N , top.begin() + N ); // v component
		thrust::copy( counter + (3 * N - Nx), counter + 3 * N , top.begin() + 2 * N); // w component

		// Bottom neighbors
		thrust::copy( counter + Nx , counter + N , bot.begin() ); // u component
		thrust::copy( counter + N + Nx , counter + 2 * N , bot.begin() + N ); // v component
		thrust::copy( counter + 2 * N + Nx , counter + 3 * N , bot.begin() + 2 * N ); // w component
		thrust::copy( counter, counter + Nx , bot.begin() + N - Nx ); // u component
		thrust::copy( counter + N, counter + N + Nx , bot.begin() + 2 * N - Nx ); // V component
		thrust::copy( counter + 2 * N, counter + 2 * N + Nx , bot.begin() + 3 * N - Nx ); // w component

		// Left neighbors
		thrust::copy( counter , counter + 3 * N - 1 , left.begin() + 1 );

		// Right neighbors
		thrust::copy( counter + 1 , counter + 3 * N , right.begin() );

		// Adjust left and right neighbors on sides
		for (int i = 0; i < Ny; ++i)
		{
			left[i * Nx] = (i + 1) * Nx - 1; // u component
			right[(i + 1) * Nx - 1] = i * Nx; // u component
			left[N + i * Nx] = N + (i + 1) * Nx - 1; // v component
			right[N + (i + 1) * Nx - 1] = N + i * Nx; // v component
			left[2 * N + i * Nx] = 2 * N + (i + 1) * Nx - 1; // w component
			right[2 * N + (i + 1) * Nx - 1] = 2 * N + i * Nx; // w component
		}
	}

	void operator() ( const state_type &x , state_type &dxdt , const value_type &t)
	{
		BOOST_LOG_TRIVIAL(debug) << "Compute dynamics for t="<<t;
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() ,
							x.begin() + N,
							x.begin() + 2 * N,
							thrust::make_constant_iterator(epsilon)
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							Pxx_top.begin(),
							Pxx_bot.begin(),
							Pxx_left.begin(),
							Pxx_right.begin()
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin(), dxdt.begin() + N, dxdt.begin() + 2 * N)
					)
			) ),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					thrust::make_zip_iterator(
						thrust::make_tuple(
							x.begin() + N,
							x.begin() + 2 * N,
							x.begin() + 3 * N,
							thrust::make_constant_iterator(epsilon)
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), top.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), top.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), top.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), bot.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), bot.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), bot.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), left.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), left.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), left.end() )
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(
							thrust::make_permutation_iterator( x.begin(), right.begin() + N ),
							thrust::make_permutation_iterator( x.begin(), right.begin() + 2 * N ),
							thrust::make_permutation_iterator( x.begin(), right.end() )
					) ) ,
					thrust::make_constant_iterator( thrust::make_tuple(cu, cv, cw, Du, Dv, Dw) ),
					thrust::make_constant_iterator( thrust::make_tuple(c1, c2, c3, c4, c5, c6, c7, c8, c9) ),
					thrust::make_constant_iterator( thrust::make_tuple(Fmax, Gmax, Hmax) ),
					thrust::make_zip_iterator(
						thrust::make_tuple(
							Pxx_top.end(),
							Pxx_bot.end(),
							Pxx_left.end(),
							Pxx_right.end()
					) ) ,
					thrust::make_zip_iterator(
						thrust::make_tuple(dxdt.begin() + N, dxdt.begin() + 2 * N, dxdt.begin() + 3 * N)
					)
			) ),
			sys_functor()
		);
	}

	const index_vector_type& get_top() const {return this->top;}
	const index_vector_type& get_bot() const {return this->bot;}
	const index_vector_type& get_left() const {return this->left;}
	const index_vector_type& get_right() const {return this->right;}
	const size_t& get_N() const {return this->N;}

private:

	const size_t N, Nx, Ny;
	const double cu, cv, cw;
	const double c1, c2, c3, c4, c5, c6, c7, c8, c9;
	const double Du, Dv, Dw;
	const double Fmax, Gmax, Hmax;
	const double epsilon;
	index_vector_type top, bot, left, right;

public:
	const state_type Pxx_top, Pxx_bot, Pxx_left, Pxx_right;
};

/**
 * \brief Compute the max length of the file names
*/
template<typename T>
size_t number_length(T &tmax, T &dt)
{
	std::ostringstream tmp;
	double int_part;
	double decimal_part = std::modf(tmax, &int_part);
	tmp << int_part + dt;
	return tmp.str().size();
}

/**
 * \brief Compute the max length of the decimal part of the file names
*/
template<typename T>
size_t number_precision(T &dt)
{
	std::ostringstream tmp;
	tmp << dt;
	return std::max(size_t(3), tmp.str().size()) - 2;
}

/**
 * \brief Define the observer used to export the results
*/
struct observer
{
	const Parameters &params;
	const size_t N;
	const size_t filename_length;
	const size_t precision;
	const state_type &Pxx_top, &Pxx_bot, &Pxx_left, &Pxx_right;
	double last_t;


	observer(
		const Parameters &params_in, const size_t &N_in,
		const state_type &Pxx_top_in,
		const state_type &Pxx_bot_in,
		const state_type &Pxx_left_in,
		const state_type &Pxx_right_in
	):
		params( params_in ), N( N_in ),
		filename_length( number_length(params.tmax, params.dt) ),
		precision( number_precision(params.dt) ),
		Pxx_top( Pxx_top_in ), Pxx_bot( Pxx_bot_in ), Pxx_left( Pxx_left_in ), Pxx_right( Pxx_right_in )
	{
		BOOST_LOG_TRIVIAL(debug) << "Observer initialization";
		last_t = -1;
	}

	template< class State >
	void operator()( const State &state , value_type t )
	{
		// TODO: use params.delta_obs to skip some exports if they are too close from each other
		if(t - last_t < params.delta_obs || t >= params.tmax - params.dt)
		{
			BOOST_LOG_TRIVIAL(debug) << "Skip export results for t="<<t;
			return;
		}
		else
		{
			last_t = t;
		}

		// Format file name (zero padding to ensure that the file are always correctly sorted)
		std::ostringstream filename;
		filename << std::fixed << std::setprecision(precision) << std::setw(filename_length) << std::setfill('0') << t << ".dat";

		BOOST_LOG_TRIVIAL(info) << "Export results for t="<<t<<" in "<<filename.str();

		// Create file
		generic::DatWriter data_file(params.result_folder + "/results/" + filename.str());

		// Write header
		data_file.write_header(std::to_string(t), params.Nx, params.Ny, "x", "y", "u", "v", "w", "P_sin_theta_top", "P_sin_theta_bot", "P_sin_theta_left", "P_sin_theta_right");

		// Write data
		// TODO: This is the slowest part of the code, try to improve it either by using binary files or by improving the iterators (but I/O are probably the main limitation)
		int num=0;
		for(
			auto i=thrust::make_zip_iterator(thrust::make_tuple(
				state.begin(),
				state.begin() + N,
				state.begin() + 2 * N,
				Pxx_top.begin(),
				Pxx_bot.begin(),
				Pxx_left.begin(),
				Pxx_right.begin()
			) );
			i != thrust::make_zip_iterator(thrust::make_tuple(
				state.begin() + N,
				state.begin() + 2 * N,
				state.begin() + 3 * N,
				Pxx_top.end(),
				Pxx_bot.end(),
				Pxx_left.end(),
				Pxx_right.end()
			) );
			++i
		)
		{
			data_file.write_row(
				get_x(num, params.Nx, params.epsilon),
				get_y(num, params.Nx, params.epsilon),
				thrust::get<0>(*i),
				thrust::get<1>(*i),
				thrust::get<2>(*i),
				thrust::get<3>(*i),
				thrust::get<4>(*i),
				thrust::get<5>(*i),
				thrust::get<6>(*i)
			);
			++num;
		}
	}
};

/**
 * \brief Random initialization
*/
void random_hex_init(host_state_type &state, const Parameters &params, host_state_type &Pxx_top, host_state_type &Pxx_bot, host_state_type &Pxx_left, host_state_type &Pxx_right)
{
	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;

	const double &epsilon = params.epsilon;
	const double &S = params.S;
	const double &P = params.P;

	BOOST_LOG_TRIVIAL(info) << "Random initialization:";
	BOOST_LOG_TRIVIAL(info) << "\t Nx = "<<Nx;
	BOOST_LOG_TRIVIAL(info) << "\t Ny = "<<Ny;
	BOOST_LOG_TRIVIAL(info) << "\t epsilon = "<<epsilon;
	BOOST_LOG_TRIVIAL(info) << "\t S = "<<S;
	BOOST_LOG_TRIVIAL(info) << "\t P = "<<P;


	// Generate hexagonal lattice
	const double hex_width(2. * S), hex_hor_space(hex_width * 3. / 4.), hex_vert_space(S * sqrt(3.));
	const size_t Nq(std::max(size_t(1), size_t((Nx * epsilon) / hex_hor_space)));
	const size_t Nr(std::max(size_t(1), size_t((Ny * epsilon) / hex_vert_space)));

	generic::Hex2dLattice hex(S, Nq, Nr);

	BOOST_LOG_TRIVIAL(debug) << "\t Hexagonal lattice generation";
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.side="<<hex.side;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.x_length="<<hex.x_length;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.y_length="<<hex.y_length;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.hex_width="<<hex.hex_width;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.hex_hor_space="<<hex.hex_hor_space;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.hex_vert_space="<<hex.hex_vert_space;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.Nq="<<hex.Nq;
	BOOST_LOG_TRIVIAL(debug) << "\t\t hex.Nr="<<hex.Nr;

	host_int_vector_type hex_colors(hex.N, 0);

    // Generate random colors
	std::random_device rd;
    std::mt19937 random_engine(rd());
    std::uniform_int_distribution<> distribution(0, 1);
	std::generate(hex_colors.begin(), hex_colors.end(), [&distribution, &random_engine]() { return distribution(random_engine); });

	BOOST_LOG_TRIVIAL(debug) << "\t Generate component values and Laplacian correction terms";

	// Initialize P * sin(theta) terms
	// Set P sin(theta) = epsilon^{-2} everywhere for all direction
	std::fill(Pxx_top.begin(), Pxx_top.end(), std::pow(epsilon, -2.0));
	std::fill(Pxx_bot.begin(), Pxx_bot.end(), std::pow(epsilon, -2.0));
	std::fill(Pxx_left.begin(), Pxx_left.end(), std::pow(epsilon, -2.0));
	std::fill(Pxx_right.begin(), Pxx_right.end(), std::pow(epsilon, -2.0));

    std::uniform_real_distribution<> perturbation(-0.1, 0.1);
	int num=0;
	for(
		auto i=thrust::make_zip_iterator(thrust::make_tuple(
			state.begin(),
			state.begin() + N,
			state.begin() + 2 * N,
			Pxx_top.begin(),
			Pxx_bot.begin(),
			Pxx_left.begin(),
			Pxx_right.begin()
		) );
		i != thrust::make_zip_iterator(thrust::make_tuple(
			state.begin() + N,
			state.begin() + 2 * N,
			state.begin() + 3 * N,
			Pxx_top.end(),
			Pxx_bot.end(),
			Pxx_left.end(),
			Pxx_right.end()
		) );
		++i
	)
	{
		// Compute coordinates of the current element
		const double x = get_x(num, Nx, epsilon);
		const double y = get_y(num, Nx, epsilon);

		// Find to which hexagon it belongs
		size_t hex_ind = hex.hex_coords_to_ind(hex.eucl_to_hex_coords(x, y));

		// Find the coordinates and color of this hexagon
		const int &color_hex = hex_colors[hex_ind];

		// Compute values of u, v, w
		const double u_init = (color_hex == 0 ? std::get<0>(black) : std::get<0>(green)) + perturbation(random_engine);
		const double v_init = (color_hex == 0 ? std::get<1>(black) : std::get<1>(green)) + perturbation(random_engine);
		const double w_init = (color_hex == 0 ? std::get<2>(black) : std::get<2>(green)) + perturbation(random_engine);

		// Set values to u, v, w
		thrust::get<0>(*i) = u_init;
		thrust::get<1>(*i) = v_init;
		thrust::get<2>(*i) = w_init;

		// Compute the Laplacian correction terms
		const std::vector<double> top_segment{x, y, x, y + epsilon};
		const std::vector<double> bot_segment{x, y, x, y - epsilon};
		const std::vector<double> left_segment{x, y, x - epsilon, y};
		const std::vector<double> right_segment{x, y, x + epsilon, y};

		auto hex_segments = hex.hex_segments(hex_ind);

		for (auto&& segment : hex_segments)
		{
			const double angle_top = hex.intersection_sin_angle(segment, top_segment);
			if(angle_top != -10.)
			{
				thrust::get<3>(*i) *= P * angle_top;
			}

			const double angle_bot = hex.intersection_sin_angle(segment, bot_segment);
			if(angle_bot != -10.)
			{
				thrust::get<4>(*i) *= P * angle_bot;
			}

			const double angle_left = hex.intersection_sin_angle(segment, left_segment);
			if(angle_left != -10.)
			{
				thrust::get<5>(*i) *= P * angle_left;
			}

			const double angle_right = hex.intersection_sin_angle(segment, right_segment);
			if(angle_right != -10.)
			{
				thrust::get<6>(*i) *= P * angle_right;
			}
		}
		++num;
	}

	// Export hexagonal lattice
	if(params.export_hex_lattice)
	{
		generic::export_hex_lattice(hex, hex_colors, params.result_folder);
	}
}

/**
 * \brief Gaussian initialization. Only used for validation.
*/
void gauss_init(host_state_type &state, const Parameters &params, host_state_type &Pxx_top, host_state_type &Pxx_bot, host_state_type &Pxx_left, host_state_type &Pxx_right)
{
	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;

	const double &epsilon = params.epsilon;
	const double &sigma = params.gauss_std;

	const double x_center = (Nx / 2.0) * epsilon;
	const double y_center = (Ny / 2.0) * epsilon;

	BOOST_LOG_TRIVIAL(info) << "Gaussian initialization:";
	BOOST_LOG_TRIVIAL(info) << "\t Nx = "<<Nx;
	BOOST_LOG_TRIVIAL(info) << "\t Ny = "<<Ny;
	BOOST_LOG_TRIVIAL(info) << "\t x_center = "<<x_center;
	BOOST_LOG_TRIVIAL(info) << "\t y_center = "<<y_center;
	BOOST_LOG_TRIVIAL(info) << "\t epsilon = "<<epsilon;
	BOOST_LOG_TRIVIAL(info) << "\t std = "<<sigma;

	int num=0;
	for(
		auto i=thrust::make_zip_iterator(thrust::make_tuple(
			state.begin(),
			state.begin() + N,
			state.begin() + 2 * N
		) );
		i != thrust::make_zip_iterator(thrust::make_tuple(
			state.begin() + N,
			state.begin() + 2 * N,
			state.begin() + 3 * N
		) );
		++i
	)
	{
		const double x = get_x(num, Nx, epsilon);
		const double y = get_y(num, Nx, epsilon);
		const double r = std::sqrt(std::pow(x - x_center, 2.) + std::pow(y - y_center, 2.));
		const double C = std::exp(-std::pow(r, 2.) / (2. * std::pow(sigma, 2.)));
		thrust::get<0>(*i) = C;
		thrust::get<1>(*i) = C;
		thrust::get<2>(*i) = C;
		++num;
	}

	// Set P sin(theta) = 1 everywhere for all direction
	std::fill(Pxx_top.begin(), Pxx_top.end(), 1.0);
	std::fill(Pxx_bot.begin(), Pxx_bot.end(), 1.0);
	std::fill(Pxx_left.begin(), Pxx_left.end(), 1.0);
	std::fill(Pxx_right.begin(), Pxx_right.end(), 1.0);
}

/**
 * \brief Export neighbors indices used to compute the laplacian. Only used for validation.
*/
void export_neighbors(const rd_dynamics &sys, const Parameters &params)
{
	BOOST_LOG_TRIVIAL(info) << "Export neighbors";
	const size_t &N=sys.get_N();

	// Create file
	generic::DatWriter data_file(params.result_folder + "/neighbors.dat");

	// Write header
	data_file.write_header("Neighbors", params.Nx, params.Ny, "x", "y", "num", "top_u", "top_v", "top_w", "bot_u", "bot_v", "bot_w", "left_u", "left_v", "left_w", "right_u", "right_v", "right_w");

	// Write data
	int num=0;
	for(
		auto i=thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin(),
						sys.get_top().begin() + N,
						sys.get_top().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() ,
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() ,
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() ,
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N
				) )
		) );
		i != thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_top().begin() + N ,
						sys.get_top().begin() + 2 * N ,
						sys.get_top().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_bot().begin() + N ,
						sys.get_bot().begin() + 2 * N ,
						sys.get_bot().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_left().begin() + N ,
						sys.get_left().begin() + 2 * N ,
						sys.get_left().end()
				) ) ,
				thrust::make_zip_iterator(
					thrust::make_tuple(
						sys.get_right().begin() + N ,
						sys.get_right().begin() + 2 * N ,
						sys.get_right().end()
				) )
		) );
		++i
	)
	{
		const double x = get_x(num, params.Nx, params.epsilon);
		const double y = get_y(num, params.Nx, params.epsilon);
		data_file.write_row(
			x, y, num,
			thrust::get<0>(thrust::get<0>(*i)), thrust::get<1>(thrust::get<0>(*i)), thrust::get<2>(thrust::get<0>(*i)),
			thrust::get<0>(thrust::get<1>(*i)), thrust::get<1>(thrust::get<1>(*i)), thrust::get<2>(thrust::get<1>(*i)),
			thrust::get<0>(thrust::get<2>(*i)), thrust::get<1>(thrust::get<2>(*i)), thrust::get<2>(thrust::get<2>(*i)),
			thrust::get<0>(thrust::get<3>(*i)), thrust::get<1>(thrust::get<3>(*i)), thrust::get<2>(thrust::get<3>(*i))
		);
		++num;
	}
}

host_state_type simulate_rd(Parameters &params)
{
	// Get values from parameters
	const double &cu=params.cu, &cv=params.cv, &cw=params.cw;
	const double &c1=params.c1, &c2=params.c2, &c3=params.c3, &c4=params.c4, &c5=params.c5, &c6=params.c6, &c7=params.c7, &c8=params.c8, &c9=params.c9;
	const double &Du=params.Du, &Dv=params.Dv, &Dw=params.Dw;
	const double &Fmax=params.Fmax, &Gmax=params.Gmax, &Hmax=params.Hmax;

	const size_t &Nx = params.Nx, &Ny = params.Ny;
	const size_t N = Nx * Ny;
	const double &epsilon = params.epsilon;
	const value_type &dt = params.dt;

	// Create vectors of data: all variables are concatenated into one vector for simplicity
	// Create initial conditions and initial values on host
	host_state_type x_host( 3 * N, 0 );
	host_state_type Pxx_top( N, 1.0 ), Pxx_bot( N, 1.0 ), Pxx_left( N, 1.0 ), Pxx_right( N, 1.0 );
	if (params.gauss_std > 0)
	{
		gauss_init(x_host, params, Pxx_top, Pxx_bot, Pxx_left, Pxx_right);
	}
	else
	{
		random_hex_init(x_host, params, Pxx_top, Pxx_bot, Pxx_left, Pxx_right);
	}

	// Copy to device
	state_type x = x_host;

	// Create stepper
	ode::runge_kutta4< state_type , value_type , state_type , value_type > stepper;

	// Create phase oscillator system function
	rd_dynamics sys(
		Nx, Ny, epsilon,
		cu, cv, cw,
		c1, c2, c3, c4, c5, c6, c7, c8, c9,
		Du, Dv, Dw,
		Fmax, Gmax, Hmax,
		Pxx_top, Pxx_bot, Pxx_left, Pxx_right
	);

	// Export neighbors
	if(params.export_neighbors)
	{
		export_neighbors(sys, params);
	}

	// Create observer
	observer obs(params, N, sys.Pxx_top, sys.Pxx_bot, sys.Pxx_left, sys.Pxx_right);

	// Integrate
	// TODO: Add stoping criteria but Boost::ODEINT does not provide an easy way to do this. I think this should be done inside the observer to interrupt the integration when the criteria is satisfied. Another solution can be to just use do_step() manually.
	integrate_const( stepper , sys , x , 0.0 , params.tmax , dt , boost::ref(obs));

	// Export results
	thrust::copy( x.begin() , x.end() , x_host.begin() );
	return x_host;
}

int main( int argc , char* argv[] )
{
	// Define and read the parameters
	Parameters params;
	params.read(argc, argv);
	std::cout<<params<<std::endl;

	// Initialize the logger
	generic::init_logger(params.log_level);

	// Create folders in which the results will be stored
	if(!boost::filesystem::is_directory(params.result_folder))
	{
		boost::filesystem::create_directories(params.result_folder);
	}
	if(!boost::filesystem::is_directory(params.result_folder + "/results"))
	{
		boost::filesystem::create_directories(params.result_folder + "/results");
	}

	// Export parameters used for the current simulation
	BOOST_LOG_TRIVIAL(info) << "Export parameters";
	params.write_parameters(params.result_folder + "/parameters_used.prm");

	// Run the simulation
	host_state_type result = simulate_rd(params);
}
